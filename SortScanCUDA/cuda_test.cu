
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void hello() {
	printf("Hello, world! This is thread %d\n", threadIdx.x);
}
int main(void) {
	hello<<<1,10>>>();
	hipDeviceSynchronize();
	cout << endl;
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	for (int device = 0; device < deviceCount; ++device) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, device);
		cout << "Device " << device << " has compute capability "
			<< deviceProp.major << "." << deviceProp.minor << endl;
	}
	return 0;
}